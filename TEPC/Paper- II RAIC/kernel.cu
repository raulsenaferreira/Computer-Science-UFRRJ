#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"

#include "hip/hip_math_constants.h"
#include "math_functions.h"
#include <stdlib.h>
#include <time.h>

#include <math.h>
#include <stdio.h>

hipError_t Calculate_KDE(const double *x, double *pdf, const int h, unsigned int size);
__device__ double GaussianKernel(double t);

__global__ void KDE(const double *x, double *pdf, const int h, const int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	int i = 0;
	double sum = 0;

	for(i = 0; i < size; i++)
	{
		sum += GaussianKernel((x[id] - x[i])/h)/h;
	}
	pdf[id] = sum/size;
}

__device__ double GaussianKernel(double t)
{
	double gaussian;

	gaussian = (1/sqrt(2 * M_PI)) * exp(-pow(t,2)/2);

	return gaussian;
}

__global__ void KDEMultivariante(const double *x, double *pdf, const int h, const int size, const int vecSize)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int i = 0, j = 0;
	double sum = 0, prod = 1;

	for(i; i < size; i++)
	{
		prod = 1;

		for(j; j < vecSize; j++)
		{
			prod *= GaussianKernel(x[id*vecSize + j] - x[i*vecSize + j]);
		}
		sum += prod;
	}

	pdf[id] = sum/size;
}

int main()
{
    const int arraySize = 30000;

    //{ 0.91, 1.01 ,0.95 ,1.13, 1.12 ,0.86 ,0.96, 1.17, 1.36 ,1.10 ,0.98 ,1.27 ,1.13 ,0.92, 1.15 ,1.26, 1.14, 0.88, 1.03, 1.00 ,0.98 ,0.94 ,1.09 ,0.92 ,1.10 ,0.95 ,1.05 ,1.05 ,1.11 ,1.15 };
	double *b = (double*)malloc(arraySize * sizeof(double));
	double c[30000] = {0};

	const double h = 1;
	int i = 0;

	for(i = 0; i < arraySize; i++)
        b[i] = ((double) rand() / (RAND_MAX)) + 0.200001;

	/*for(i = 0; i < arraySize; i++)
        c[i] = 0.0;*/

    clock_t init,fin;
    double time_spent;

	printf("%f %f %f\n", b[0], b[arraySize/2], b[arraySize - 1]);

    init = clock();
    // Add vectors in parallel.
    hipError_t cudaStatus = Calculate_KDE(b, c, h, arraySize);
	fin = clock();
    time_spent = (double)(fin - init) / CLOCKS_PER_SEC;

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	/* int k;
	for(k = 0; k < 30; k++)
        printf("%f \n",c[k]);*/

	printf("%f %f %f\n", c[0], c[arraySize/2], c[arraySize - 1]);
	printf("Executado em %f segundos\n", time_spent);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t Calculate_KDE(const double *x, double *pdf, const int h, unsigned int size)
{
	double *observation = 0;
    double *result = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&result, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&observation, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(observation, x, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 1 failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(result, pdf, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy  2 failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    KDE<<<128, 256>>>(observation, result, h, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pdf, result, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 3 failed!");
        goto Error;
    }

Error:
    hipFree(result);
    hipFree(observation);
    
    return cudaStatus;
}
