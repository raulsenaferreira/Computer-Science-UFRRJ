#include <stdio.h>
#include <hip/hip_runtime.h>

typedef struct biscoito{
	int indiceBloco;
	int indiceThread;
	int global;
} Biscoito;

__global__ void somaMatrizGPU(int *d_vetA, int *d_vetB, int *d_vetC, Biscoito *bis){
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	bis[id].indiceThread = threadIdx.x;
	bis[id].indiceBloco = blockDim.x;
	bis[id].global = id;

	d_vetC[id] = d_vetA[id] + d_vetB[id];
}

int main(){

	int h_Size = 16;
	int i = 0;
	int h_vetA[16]={1,2,3,4,5,6,7,8,9,10, 11, 12, 13, 14, 15, 16};
	int h_vetB[16]={1,2,3,4,5,6,7,8,9,10, 11, 12, 13, 14, 15, 16};
	int h_vetC[16];	
	int *d_vetA;
	int *d_vetB;	
	int *d_vetC;
	Biscoito *biscoito = (Biscoito *) malloc(16 * sizeof(Biscoito)) ;
	Biscoito *bis;

	hipDeviceReset();
	hipMalloc((void**) &d_vetA, h_Size * sizeof(int));
	hipMalloc((void**) &d_vetB, h_Size * sizeof(int));
	hipMalloc((void**) &d_vetC, h_Size * sizeof(int));
	hipMalloc((void**) &bis, 16 * sizeof(Biscoito));

	hipMemcpy(d_vetA, h_vetA, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	hipMemcpy(d_vetB, h_vetB, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	hipMemcpy(d_vetC, h_vetC, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	hipMemcpy(bis, biscoito, 16 * sizeof(Biscoito),  hipMemcpyHostToDevice);
	
	somaMatrizGPU<<<8, 2>>>(d_vetA, d_vetB, d_vetC, bis);

	hipDeviceSynchronize();
	
	hipMemcpy(h_vetC, d_vetC, h_Size * sizeof(int),  hipMemcpyDeviceToHost);
	hipMemcpy(biscoito, bis, 16 * sizeof(Biscoito),  hipMemcpyDeviceToHost);

	/*for(i=0; i < h_Size; i++){
		printf("%d, ", h_vetC[i]);	
	}*/
	
	for(i=0; i < 2; i++){
		printf("%d, %d, %d -- \n", biscoito[i].indiceBloco, biscoito[i].indiceThread, biscoito[i].global);	
	}
	
	hipFree(d_vetA);
	hipFree(d_vetB);
	hipFree(d_vetC);
	hipFree(bis);

	return 0;
		
}
