#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void somaMatrizGPU(int *d_vetA, int *d_vetB, int *d_vetC){
	int id = blockDim.x * blockIdx.x + threadIdx.x;	
	
	d_vetC[id] = d_vetA[id] + d_vetB[id];
}

int main(){

	int h_Size = 16;
	int i = 0;
	int h_vetA[16]={1,2,3,4,5,6,7,8,9,10, 11, 12, 13, 14, 15, 16};
	int h_vetB[16]={1,2,3,4,5,6,7,8,9,10, 11, 12, 13, 14, 15, 16};
	int h_vetC[16];	
	int *d_vetA;
	int *d_vetB;	
	int *d_vetC;

	hipDeviceReset();
	hipMalloc((void**) &d_vetA, h_Size * sizeof(int));
	hipMalloc((void**) &d_vetB, h_Size * sizeof(int));
	hipMalloc((void**) &d_vetC, h_Size * sizeof(int));

	hipMemcpy(d_vetA, h_vetA, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	hipMemcpy(d_vetB, h_vetB, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	hipMemcpy(d_vetC, h_vetC, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	
	somaMatrizGPU<<<8, 2>>>(d_vetA, d_vetB, d_vetC);

	hipDeviceSynchronize();

	
	hipMemcpy(h_vetC, d_vetC, h_Size * sizeof(int),  hipMemcpyDeviceToHost);

	for(i=0; i < h_Size; i++){
		printf("%d", h_vetC[i]);	
	}
	
	hipFree(d_VetA);
	hipFree(d_VetB);
	hipFree(d_VetC);

	return 0;
		
}
