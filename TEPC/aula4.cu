#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void somaMatrizGPU(int *d_vetA, int passo){

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(id < passo)
		d_vetA[id] += d_vetA[id+passo];
}

int main(){
	
	int h_Size = 16;
	int j, i = 0;
	int h_vetA[16]={1,2,3,4,5,6,7,8,9,10, 11, 12, 13, 14, 15, 16};
	int *d_vetA;
	int passo, indice;
	int block = h_Size;

	passo = h_Size;

	hipDeviceReset();
	hipMalloc((void**) &d_vetA, h_Size * sizeof(int));

	hipMemcpy(d_vetA, h_vetA, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	
	for(i = 0; i < (int) ceil(log(h_Size) / log(2)) ; i++){
		
		passo = passo / 2;

		somaMatrizGPU<<<8, 2>>>(d_vetA, passo);

		hipMemcpy(h_vetA, d_vetA, h_Size * sizeof(int),  hipMemcpyDeviceToHost);

		for(j=0; j < h_Size; j++){
			printf("%d, ", h_vetA[j]);	
		}
		printf("\n");
	}
	
	hipDeviceSynchronize();
	
	hipFree(d_vetA);

	return 0;
		
}
