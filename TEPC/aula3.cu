#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void somaMatrizGPU(int *d_vetA, int indice, int passo){

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	
	if((id % indice) == 0)

		d_vetA[id] += d_vetA[id+passo];
}

int main(){
	
	int h_Size = 16;
	int j, i = 0;
	int h_vetA[16]={1,2,3,4,5,6,7,8,9,10, 11, 12, 13, 14, 15, 16};
	int *d_vetA;
	int passo, indice;
	int block = h_Size;

	hipDeviceReset();
	hipMalloc((void**) &d_vetA, h_Size * sizeof(int));

	hipMemcpy(d_vetA, h_vetA, h_Size * sizeof(int),  hipMemcpyHostToDevice);
	
	for(i = 0; i < 4; i++){
		
		indice = pow(2, i+1);
		passo = pow(2, i);

		somaMatrizGPU<<<8, 2>>>(d_vetA, indice, passo);

		hipMemcpy(h_vetA, d_vetA, h_Size * sizeof(int),  hipMemcpyDeviceToHost);

		for(j=0; j < h_Size; j++){
			printf("%d, ", h_vetA[j]);	
		}
		printf("\n");
	}
	
	hipDeviceSynchronize();
	
	hipFree(d_vetA);

	return 0;
		
}
