#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"

#include "hip/hip_math_constants.h"
#include "math_functions.h"

#include <math.h>
#include <stdio.h>

hipError_t Calculate_KDE(const double *x, double *pdf, const int h, unsigned int size);
__device__ double GaussianKernel(double t);

__global__ void KDE(const double *x, double *pdf, const int h, const int size)
{
    int id = threadIdx.x;
	int i = 0;
	double sum = 0;

	for(i = 0; i < size; i++)
	{
		sum += GaussianKernel((x[id] - x[i])/h)/h;
	}
	pdf[id] = sum/size;
}

__device__ double GaussianKernel(double t)
{
	double gaussian;

	gaussian = (1/sqrt(2 * M_PI)) * exp(-pow(t,2)/2);

	return gaussian;
}

int main()
{
    const int arraySize = 30;
    const double b[arraySize] = { 0.91, 1.01 ,0.95 ,1.13, 1.12 , 0.86 ,0.96, 1.17, 1.36 ,1.10 ,0.98 ,1.27 ,1.13 ,0.92, 1.15 ,1.26, 1.14, 0.88, 1.03, 1.00 ,0.98 ,0.94 ,1.09 ,0.92 ,1.10 ,0.95 ,1.05 ,1.05 ,1.11 ,1.15 };
    double c[arraySize] = { 0 };
	const int h = 1;

    // Add vectors in parallel.
    hipError_t cudaStatus = Calculate_KDE(b, c, h, arraySize);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	 int k;
	for(k = 0; k < 30; k++)
        printf("%f \n",c[k]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t Calculate_KDE(const double *x, double *pdf, const int h, unsigned int size)
{
	double *observation = 0;
    double *result = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&result, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&observation, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(observation, x, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 1 failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(result, pdf, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy  2 failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    KDE<<<1, size>>>(observation, result, h, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pdf, result, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 3 failed!");
        goto Error;
    }

Error:
    hipFree(result);
    hipFree(observation);
    
    return cudaStatus;
}
